#include "hip/hip_runtime.h"
#include "../include/arrays.h"

__global__ void array_squares_sum_kernel(float *data, float *dest, size_t len )
{
	for(int i=threadIdx.x; i < len; i += blockDim.x )
		dest[threadIdx.x] += data[i]*data[i];
}

float array_squares_sum(float *data, size_t len )
{
	float sqsum = 0, *device_sqsums, host_sqsums[64];
	hipMalloc(&device_sqsums, 64*sizeof(float) );
	array_squares_sum_kernel<<<1,256>>>(data, device_sqsums, len );
	hipMemcpy(host_sqsums, device_sqsums, 64*sizeof(float), hipMemcpyDeviceToHost );
	hipFree(device_sqsums);
	for(size_t i=0; i < 64; i++ )
		sqsum += host_sqsums[i];
	return sqsum;
}

__global__ void array_sum_kernel(float *data, float *dest, size_t len )
{
	for(int i=threadIdx.x; i < len; i += blockDim.x )
		dest[threadIdx.x] += data[i];
}

float array_sum(float *data, size_t len )
{
	float sum = 0, *device_sums, host_sums[64];
	hipMalloc(&device_sums, 64*sizeof(float) );
	array_sum_kernel<<<1,256>>>(data, device_sums, len );
	hipMemcpy(host_sums, device_sums, 64*sizeof(float), hipMemcpyDeviceToHost );
	hipFree(device_sums);
	for(size_t i=0; i < 64; i++ )
		sum += host_sums[i];
	return sum;
}

__global__ void array_scale_up_kernel(float *data, float factor, size_t len )
{
	for(int i=threadIdx.x; i < len; i += blockDim.x )
		data[i]*=factor;
}

void array_scale_up(float *data, float factor, size_t len )
{
	/*for(size_t i=0; i < len; i++ )
		data[i]*=factor;*/
	array_scale_up_kernel<<<1,256>>>(data, factor, len );
}

__global__ void array_scale_down_kernel(float *data, float factor, size_t len )
{
	for(int i=threadIdx.x; i < len; i += blockDim.x )
		data[i]/=factor;
}

void array_scale_down(float *data, float factor, size_t len )
{
	/*for(size_t i=0; i < len; i++ )
		data[i] /= factor;*/
	array_scale_down_kernel<<<1,256>>>(data, factor, len );
}

__global__ void array_step_kernel(float *dest, float *src, float factor, size_t len )
{
	for(int i=threadIdx.x; i < len; i += blockDim.x )
		dest[i]+=src[i]*factor;
}

void array_step(float *dest, float *src, float factor, size_t len )
{
	/*for(size_t i=0; i < len; i++ )
		dest[i] += src[i]*factor;*/
	array_step_kernel<<<1,256>>>(dest, src, factor, len );
}

__global__ void array_abs_kernel(float *dest, size_t len )
{
	for(int i=threadIdx.x; i < len; i += blockDim.x )
		dest[i] = fabs(dest[i]);
}

void array_abs(float *data, size_t len )
{
	/*for(size_t i=0; i < len; i++ )
		data[i] = fabs(data[i]);*/
	array_abs_kernel<<<1,256>>>(data, len );
}

void normalize(float *data, size_t len )
{
	/*float dist=0;
	for(size_t i=0; i < len; i++ )
		dist += data[i]*data[i];
	dist = sqrt(dist);
	for(size_t i=0; i < len; i++ )
		data[i] /= dist;*/
	float dist = array_squares_sum(data, len );
	dist = sqrt(dist);
	array_scale_down(data, dist, len );
}

