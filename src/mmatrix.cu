#include "hip/hip_runtime.h"
#include "../include/mmatrix.h"

void matrix_alloc(matrix_ut *m )
{ hipMalloc((void**)&m->data, sizeof(float)*m->size[0] ); }

void mmatrix_alloc(mmatrix_ut *mm )
{ hipMalloc((void **)&mm->data, sizeof(float)*mm->size[0] ); }

void copy_matrix_size(matrix_ut *src, matrix_ut *dest )
{
	dest->size[0] = src->size[0];
	dest->size[1] = src->size[1];
}

void copy_mmatrix_size(mmatrix_ut *src, mmatrix_ut *dest )
{
	for(size_t i=0; i < 4; i++ )
		dest->size[i] = src->size[i];
}

__global__ void hadamard_matrix_matrix_kernel(float *m0, float *m1, float *m_out, size_t size )
{
	for(int i=threadIdx.x; i < size; i += blockDim.x )
		m_out[i] = m0[i] * m1[i];
}

void hadamard_matrix_matrix(matrix_ut *m0, matrix_ut *m1, matrix_ut *m_out )
{
	m_out->size[0] = m0->size[0]; m_out->size[1] = m0->size[1];
	hadamard_matrix_matrix_kernel<<<1,256>>>(m0->data, m1->data, m_out->data, m0->size[0] );
}

void set_mul_matrix_matrix_size(matrix_ut *m0, matrix_ut *m1, matrix_ut *m_out )
{
	m_out->size[1] = m1->size[1];
	m_out->size[0] = m0->size[0]/m0->size[1] * m_out->size[1];
}

__global__ void mul_matrix_matrix_kernel(float *m0, float *m1, float *m_out, size_t m, size_t n, size_t p )
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if( row < m && col < p )
	{
		m_out[row*p+col] = 0;
		for(int vert_m=0, horiz_m=0; horiz_m < n; horiz_m++, vert_m+=p )
			m_out[row*p+col] += m0[row*n+horiz_m] * m1[vert_m+col];
        }

}

void mul_matrix_matrix(matrix_ut *m0, matrix_ut *m1, matrix_ut *m_out )
{
	/*for(size_t k=0; k < m_out->size[0]; k++ )
		m_out->data[k] = 0;
	for(size_t i_in=0,i_out=0; i_in < m0->size[0]; i_in+=m0->size[1],i_out+=m_out->size[1] )
		for(size_t j_in=0,j_out=0; j_in < m1->size[1]; j_in++,j_out++ )
			for(size_t k0=0,k1=0; k0 < m0->size[1]; k0++,k1+=m_out->size[1] )
				m_out->data[i_out+j_out] += m0->data[i_in+k0]*m1->data[k1+j_in];*/
	size_t m = m0->size[0] / m0->size[1], n = m0->size[1], p = m1->size[1];
	mul_matrix_matrix_kernel<<<1,dim3(m,p,1)>>>(m0->data, m1->data, m_out->data, m, n, p );	
}

void set_mul_mmatrix_matrix_size(mmatrix_ut *mm0, matrix_ut *m1, matrix_ut *mm_out )
{
	mm_out->size[3] = mm0->size[3]; mm_out->size[2] = mm0->size[2];
	mm_out->size[1] = mm_out->size[2] * m1->size[1]; mm_out->size[0] = mm_out->size[1] * mm0->size[0] / mm0->size[1];
}

__global__ void mul_mmatrix_matrix_kernel(float *mm0, float *m1, float *mm_out, size_t m, size_t n, size_t p, size_t l )
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if( row < m && col < p )
	{
		for(int i=threadIdx.z; i < l; i += blockDim.z ) 
			mm_out[row*p+col*l+i] = 0;
		for(int vert_m=0, horiz_m=0; horiz_m < n; horiz_m++, vert_m+=p )
			for(int i=threadIdx.z; i < l; i += blockDim.z )
				mm_out[row*p+col*l+i] += mm0[row*n*l+horiz_m*l+i] * m1[vert_m+col];
        }

}

void mul_mmatrix_matrix(mmatrix_ut *mm0, matrix_ut *m1, mmatrix_ut *mm_out )
{
	/*for(size_t m=0; m < mm_out->size[0]; m++ )
		mm_out->data[m] = 0;
	for(size_t i_in=0, i_out=0; i_in < mm0->size[0]; i_in+=mm0->size[1],i_out+=mm_out->size[1] )
		for(size_t j=0,j_out=0; j < m1->size[1]; j++,j_out+=mm_out->size[2] )
		{
			for(size_t k=0,k_in=0; k < m1->size[0]; k+=m1->size[1],k_in+=mm0->size[2] )
			{
				for(size_t m=0; m < mm_out->size[2]; m++ )
					mm_out->data[i_out+j_out+m] += mm0->data[i_in+k_in+m]*m1->data[k+j];
			}
		}*/
	size_t m = mm0->size[0] / mm0->size[1], n = mm0->size[1]/mm0->size[2], p = m1->size[1], l = mm0->size[2];
	mul_mmatrix_matrix_kernel<<<1,dim3(m,p,4)>>>(mm0->data, m1->data, mm_out->data, m, n, p, l );
}

void set_mul_matrix_mmatrix(matrix_ut *m0, mmatrix_ut *mm1, mmatrix_ut *mm_out )
{
	mm_out->size[3] = mm1->size[3]; mm_out->size[2] = mm1->size[2]; mm_out->size[1] = mm1->size[1];
	mm_out->size[0] = mm_out->size[1] * m0->size[0] / m0->size[1];
}	

__global__ void mul_matrix_mmatrix_kernel(float *m0, float *mm1, float *mm_out, size_t m, size_t n, size_t p, size_t l )
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if( row < m && col < p )
	{
		for(int i=threadIdx.z; i < l; i += blockDim.z ) 
			mm_out[row*p+col*l+i] = 0;
		for(int vert_m=0, horiz_m=0; horiz_m < n; horiz_m++, vert_m+=p*l )
			for(int i=threadIdx.z; i < l; i += blockDim.z )
				mm_out[row*p*l+col*l+i] += m0[row*n+horiz_m] * mm1[vert_m+col*l+i];
        }

}

void mul_matrix_mmatrix(matrix_ut *m0, mmatrix_ut *mm1, mmatrix_ut *mm_out ) 
{
		/*for(size_t m=0; m < mm_out->size[0]; m++ )
			mm_out->data[m] = 0;
		for(size_t i=0, i_out=0; i < m0->size[0]; i+=m0->size[1],i_out+=mm_out->size[1] )
			for(size_t j_in=0, j_out=0; j_in < mm1->size[1]; j_in+=mm1->size[2],j_out+=mm_out->size[2] )
			{
				for(size_t k=0, k_in=0; k < m0->size[1]; k++,k_in+=mm1->size[1] )
					for(size_t m=0; m < mm_out->size[2]; m++ )
						mm_out->data[i_out+j_out+m] += m0->data[i+k]*mm1->data[j_in+k_in+m];
			} */
	size_t m = m0->size[0] / m0->size[1], n = m0->size[1], p = mm1->size[1]/mm1->size[2], l = mm1->size[2];
	mul_matrix_mmatrix_kernel<<<1,dim3(m,p,4)>>>(m0->data, mm1->data, mm_out->data, m, n, p, l );
}

__global__ void hadamard_mmatrix_matrix_kernel(float *mm0, float *m1, float *mm_out, size_t n, size_t m, size_t l )
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	for(size_t i=threadIdx.z; i < l; i +=blockDim.z )
		mm_out[(row*n+col)*l+i] = mm0[(row*n+col)*l+i] * m1[row*n+col];	

}

void hadamard_mmatrix_matrix(mmatrix_ut *mm0, matrix_ut *m1, mmatrix_ut *mm_out )
{
	/*for(size_t i=0,i_in=0; i < m1->size[0]; i+=m1->size[1],i_in+=mm0->size[1] )
       		for(size_t j=0,j_in=0; j < m1->size[1]; j++,j_in+=mm0->size[2] )
			for(size_t m=0; m < mm0->size[2]; m++ )
				mm_out->data[i_in+j_in+m] = mm0->data[i_in+j_in+m]*m1->data[i+j];*/
	size_t m = m1->size[0]/m1->size[1], n = m1->size[1], l = mm0->size[2];
	hadamard_mmatrix_matrix_kernel<<<1,dim3(m,n,4)>>>(mm0->data, m1->data, mm_out->data, m, n, l );
}

void set_mul_lodelta_matrix(matrix_ut *m0, matrix_ut *m1, mmatrix_ut *mm_out )
{
	mm_out->size[3] = m0->size[1]; mm_out->size[2] = m0->size[0]; mm_out->size[1] = m1->size[1]*mm_out->size[2];
	mm_out->size[0] = m0->size[0] / m0->size[1] * mm_out->size[1];
}

__global__ void mul_lodelta_matrix_kernel(float *m0, float *m1, float *mm_out, size_t m, size_t n, size_t p )
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;

	if( row < m && col < p )
	{
		//for(int i=threadIdx.z; i < l; i += blockDim.z ) 
		//	mm_out[row*p+col*l+i] = 0;
		for(int vert_m=0, horiz_m=0; horiz_m < n; horiz_m++, vert_m+=p )
			for(int i=threadIdx.z; i < n; i += blockDim.z )
				mm_out[row*(p*m*n+n)+col*m*n+i] += ( (col==i) - m0[row*n+horiz_m] ) * m1[vert_m+col];
        }

}

void mul_lodelta_matrix(matrix_ut *m0, matrix_ut *m1, mmatrix_ut *mm_out )
{
	/*for(size_t m=0; m < mm_out->size[0]; m++ )
		mm_out->data[m] = 0;
	for(size_t i_in=0,i_out=0,m=0; i_in < m0->size[0]; i_in+=m0->size[1],i_out+=mm_out->size[1],m+=mm_out->size[3] )
		for(size_t j_in=0,j_out=0; j_in < m1->size[1]; j_in++,j_out+=mm_out->size[2] )
			for(size_t k0=0,k1=0; k0 < m0->size[1]; k0++,k1+=m1->size[1] )
				for(size_t l=0; l < m0->size[1]; l++ )
					mm_out->data[i_out+j_out+m+l] += ( k0==l ? 1-m0->data[i_in+k0] : -m0->data[i_in+k0] )*m1->data[k1+j_in];*/
	set_mmatrix_scalar(mm_out, 0 );
	size_t m = m0->size[0]/m0->size[1], n = m0->size[1], p = m1->size[1];
	mul_lodelta_matrix_kernel<<<1,dim3(m,p,4)>>>(m0->data, m1->data, mm_out->data, m, n, p );
}

__global__ void sum_kernel(float *m0, float *m1, float *m_out, size_t size )
{
	for(int i=threadIdx.x; i < size; i+=blockDim.x )
		m_out[i] = m0[i] + m1[i];
}

void sum_matrix_matrix(matrix_ut *m0, matrix_ut *m1, matrix_ut *m_out )
{
	/*for(size_t m=0; m < m0->size[0]; m++ )
		m_out->data[m] = m0->data[m] + m1->data[m];*/
	sum_kernel<<<1,256>>>(m0->data, m1->data, m_out->data, m0->size[0] );
}

void sum_mmatrix_mmatrix(mmatrix_ut *mm0, mmatrix_ut *mm1, mmatrix_ut *mm_out )
{
	/*for(size_t m=0; m < mm0->size[0]; m++ )
		mm_out->data[m] = mm0->data[m] + mm1->data[m];*/
	sum_kernel<<<1,256>>>(mm0->data, mm1->data, mm_out->data, mm0->size[0] );
}

__global__ void mul_scalar_kernel(float *m0, float *m_out, float k, size_t size )
{
	for(int i=threadIdx.x; i < size; i+=blockDim.x )
		m_out[i] = m0[i] * k;
}

void mul_matrix_scalar(matrix_ut *m0, float k, matrix_ut *m_out )
{
	/*for(size_t m=0; m < m0->size[0]; m++ )
		m_out->data[m] = m0->data[m]*k;*/
	mul_scalar_kernel<<<1,256>>>(m0->data, m_out->data, k, m0->size[0] );
}

void mul_mmatrix_scalar(mmatrix_ut *mm0, float k, mmatrix_ut *mm_out )
{
	/*for(size_t m=0; m < mm0->size[0]; m++ )
		mm_out->data[m] = mm0->data[m]*k;*/
	mul_scalar_kernel<<<1,256>>>(mm0->data, mm_out->data, k, mm0->size[0] );
}

__global__ void sum_scalar_kernel(float *m0, float f, float *m_out, size_t size )
{
	for(int i=threadIdx.x; i < size; i+=blockDim.x )
		m_out[i] = m0[i] + f;
}

void sum_matrix_scalar(matrix_ut *m0, float f, matrix_ut *m_out )
{
	/*for(size_t m=0; m < m0->size[0]; m++ )
		m_out->data[m] = m0->data[m] + f;*/
	sum_scalar_kernel<<<1,256>>>(m0->data, f, m_out->data, m0->size[0] );
}

void sum_mmatrix_scalar(mmatrix_ut *mm0, float f, mmatrix_ut *mm_out )
{
	/*for(size_t m=0; m < mm0->size[0]; m++ )
		mm_out->data[m] = mm0->data[m]+f;*/
	sum_scalar_kernel<<<1,256>>>(mm0->data, f, mm_out->data, mm0->size[0] );
}

__global__ void set_scalar_kernel(float *m0, float f, size_t size )
{
	for(int i=threadIdx.x; i < size; i+=blockDim.x )
		m0[i] = f;
}	

void set_matrix_scalar(matrix_ut *m0, float f )
{ set_scalar_kernel<<<1,256>>>(m0->data, f, m0->size[0] ); }

void set_mmatrix_scalar(mmatrix_ut *mm0, float f )
{ set_scalar_kernel<<<1,256>>>(mm0->data, f, mm0->size[0] ); }

void print_mmatrix(mmatrix_ut *mm0, char *name )
{
	printf("%s:\n", name );
	for(size_t i=0; i < mm0->size[0]; i+= mm0->size[1] )
	{
		for(size_t m=0; m < mm0->size[2]; m+=mm0->size[3] )
		{
			for(size_t j=0; j < mm0->size[1]; j+=mm0->size[2] )
			{
				for(size_t n=0; n < mm0->size[3]; n++ )
					printf("%f ", mm0->data[i+j+m+n] );
				putchar(' ');
			}
			putchar('\n');
		}
		putchar('\n');
	}
	putchar('\n');
}

void print_matrix(matrix_ut *m0, char *name )
{
	printf("%s:\n", name );
	for(size_t i=0; i < m0->size[0]; i+=m0->size[1] )
	{
		for(size_t j=0; j < m0->size[1]; j++ )
			printf("%f ", m0->data[i+j] );
		putchar('\n');
	}
	putchar('\n');
}
